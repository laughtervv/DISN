
#include <hip/hip_runtime.h>
__global__ void approxmatch(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,float * __restrict__ match,float * temp){
	float * remainL=temp+blockIdx.x*(n+m)*2, * remainR=temp+blockIdx.x*(n+m)*2+n,*ratioL=temp+blockIdx.x*(n+m)*2+n+m,*ratioR=temp+blockIdx.x*(n+m)*2+n+m+n;
	float multiL,multiR;
	if (n>=m){
		multiL=1;
		multiR=n/m;
	}else{
		multiL=m/n;
		multiR=1;
	}
	const int Block=1024;
	__shared__ float buf[Block*4];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x;j<n*m;j+=blockDim.x)
			match[i*n*m+j]=0;
		for (int j=threadIdx.x;j<n;j+=blockDim.x)
			remainL[j]=multiL;
		for (int j=threadIdx.x;j<m;j+=blockDim.x)
			remainR[j]=multiR;
		__syncthreads();
		for (int j=7;j>=-2;j--){
			float level=-powf(4.0f,j);
			if (j==-2){
				level=0;
			}
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				float x1=0,y1=0,z1=0;
				if (k<n){
					x1=xyz1[i*n*3+k*3+0];
					y1=xyz1[i*n*3+k*3+1];
					z1=xyz1[i*n*3+k*3+2];
				}
				float suml=1e-9f;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						float x2=xyz2[i*m*3+l0*3+l*3+0];
						float y2=xyz2[i*m*3+l0*3+l*3+1];
						float z2=xyz2[i*m*3+l0*3+l*3+2];
						buf[l*4+0]=x2;
						buf[l*4+1]=y2;
						buf[l*4+2]=z2;
						buf[l*4+3]=remainR[l0+l];
					}
					__syncthreads();
					for (int l=0;l<lend;l++){
						float x2=buf[l*4+0];
						float y2=buf[l*4+1];
						float z2=buf[l*4+2];
						float d=level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1));
						float w=__expf(d)*buf[l*4+3];
						suml+=w;
					}
					__syncthreads();
				}
				if (k<n)
					ratioL[k]=remainL[k]/suml;
			}
			/*for (int k=threadIdx.x;k<n;k+=gridDim.x){
				float x1=xyz1[i*n*3+k*3+0];
				float y1=xyz1[i*n*3+k*3+1];
				float z1=xyz1[i*n*3+k*3+2];
				float suml=1e-9f;
				for (int l=0;l<m;l++){
					float x2=xyz2[i*m*3+l*3+0];
					float y2=xyz2[i*m*3+l*3+1];
					float z2=xyz2[i*m*3+l*3+2];
					float w=expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*remainR[l];
					suml+=w;
				}
				ratioL[k]=remainL[k]/suml;
			}*/
			__syncthreads();
			for (int l0=0;l0<m;l0+=blockDim.x){
				int l=l0+threadIdx.x;
				float x2=0,y2=0,z2=0;
				if (l<m){
					x2=xyz2[i*m*3+l*3+0];
					y2=xyz2[i*m*3+l*3+1];
					z2=xyz2[i*m*3+l*3+2];
				}
				float sumr=0;
				for (int k0=0;k0<n;k0+=Block){
					int kend=min(n,k0+Block)-k0;
					for (int k=threadIdx.x;k<kend;k+=blockDim.x){
						buf[k*4+0]=xyz1[i*n*3+k0*3+k*3+0];
						buf[k*4+1]=xyz1[i*n*3+k0*3+k*3+1];
						buf[k*4+2]=xyz1[i*n*3+k0*3+k*3+2];
						buf[k*4+3]=ratioL[k0+k];
					}
					__syncthreads();
					for (int k=0;k<kend;k++){
						float x1=buf[k*4+0];
						float y1=buf[k*4+1];
						float z1=buf[k*4+2];
						float w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*buf[k*4+3];
						sumr+=w;
					}
					__syncthreads();
				}
				if (l<m){
					sumr*=remainR[l];
					float consumption=fminf(remainR[l]/(sumr+1e-9f),1.0f);
					ratioR[l]=consumption*remainR[l];
					remainR[l]=fmaxf(0.0f,remainR[l]-sumr);
				}
			}
			/*for (int l=threadIdx.x;l<m;l+=blockDim.x){
				float x2=xyz2[i*m*3+l*3+0];
				float y2=xyz2[i*m*3+l*3+1];
				float z2=xyz2[i*m*3+l*3+2];
				float sumr=0;
				for (int k=0;k<n;k++){
					float x1=xyz1[i*n*3+k*3+0];
					float y1=xyz1[i*n*3+k*3+1];
					float z1=xyz1[i*n*3+k*3+2];
					float w=expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*ratioL[k];
					sumr+=w;
				}
				sumr*=remainR[l];
				float consumption=fminf(remainR[l]/(sumr+1e-9f),1.0f);
				ratioR[l]=consumption*remainR[l];
				remainR[l]=fmaxf(0.0f,remainR[l]-sumr);
			}*/
			__syncthreads();
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				float x1=0,y1=0,z1=0;
				if (k<n){
					x1=xyz1[i*n*3+k*3+0];
					y1=xyz1[i*n*3+k*3+1];
					z1=xyz1[i*n*3+k*3+2];
				}
				float suml=0;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						buf[l*4+0]=xyz2[i*m*3+l0*3+l*3+0];
						buf[l*4+1]=xyz2[i*m*3+l0*3+l*3+1];
						buf[l*4+2]=xyz2[i*m*3+l0*3+l*3+2];
						buf[l*4+3]=ratioR[l0+l];
					}
					__syncthreads();
					float rl=ratioL[k];
					if (k<n){
						for (int l=0;l<lend;l++){
							float x2=buf[l*4+0];
							float y2=buf[l*4+1];
							float z2=buf[l*4+2];
							float w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*rl*buf[l*4+3];
							match[i*n*m+(l0+l)*n+k]+=w;
							suml+=w;
						}
					}
					__syncthreads();
				}
				if (k<n)
					remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}
			/*for (int k=threadIdx.x;k<n;k+=blockDim.x){
				float x1=xyz1[i*n*3+k*3+0];
				float y1=xyz1[i*n*3+k*3+1];
				float z1=xyz1[i*n*3+k*3+2];
				float suml=0;
				for (int l=0;l<m;l++){
					float x2=xyz2[i*m*3+l*3+0];
					float y2=xyz2[i*m*3+l*3+1];
					float z2=xyz2[i*m*3+l*3+2];
					float w=expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*ratioL[k]*ratioR[l];
					match[i*n*m+l*n+k]+=w;
					suml+=w;
				}
				remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}*/
			__syncthreads();
		}
	}
}
void approxmatchLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,float * match,float * temp){
	approxmatch<<<32,512>>>(b,n,m,xyz1,xyz2,match,temp);
}
__global__ void matchcost(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,const float * __restrict__ match,float * __restrict__ out){
	__shared__ float allsum[512];
	const int Block=1024;
	__shared__ float buf[Block*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		float subsum=0;
		for (int k0=0;k0<n;k0+=blockDim.x){
			int k=k0+threadIdx.x;
			float x1=0,y1=0,z1=0;
			if (k<n){
				x1=xyz1[i*n*3+k*3+0];
				y1=xyz1[i*n*3+k*3+1];
				z1=xyz1[i*n*3+k*3+2];
			}
			for (int l0=0;l0<m;l0+=Block){
				int lend=min(m,l0+Block)-l0;
				for (int l=threadIdx.x;l<lend*3;l+=blockDim.x)
					buf[l]=xyz2[i*m*3+l0*3+l];
				__syncthreads();
				if (k<n){
					for (int l=0;l<lend;l++){
						float x2=buf[l*3+0];
						float y2=buf[l*3+1];
						float z2=buf[l*3+2];
						float d=sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1));
						subsum+=d*match[i*n*m+(l0+l)*n+k];
					}
				}
				__syncthreads();
			}
		}
		allsum[threadIdx.x]=subsum;
		for (int j=1;j<blockDim.x;j<<=1){
			__syncthreads();
			if ((threadIdx.x&j)==0 && threadIdx.x+j<blockDim.x){
				allsum[threadIdx.x]+=allsum[threadIdx.x+j];
			}
		}
		if (threadIdx.x==0)
			out[i]=allsum[0];
		__syncthreads();
	}
}
void matchcostLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,const float * match,float * out){
	matchcost<<<32,512>>>(b,n,m,xyz1,xyz2,match,out);
}
__global__ void matchcostgrad2(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,const float * __restrict__ match,float * __restrict__ grad2){
	__shared__ float sum_grad[256*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		int kbeg=m*blockIdx.y/gridDim.y;
		int kend=m*(blockIdx.y+1)/gridDim.y;
		for (int k=kbeg;k<kend;k++){
			float x2=xyz2[(i*m+k)*3+0];
			float y2=xyz2[(i*m+k)*3+1];
			float z2=xyz2[(i*m+k)*3+2];
			float subsumx=0,subsumy=0,subsumz=0;
			for (int j=threadIdx.x;j<n;j+=blockDim.x){
				float x1=x2-xyz1[(i*n+j)*3+0];
				float y1=y2-xyz1[(i*n+j)*3+1];
				float z1=z2-xyz1[(i*n+j)*3+2];
				float d=match[i*n*m+k*n+j]*rsqrtf(fmaxf(x1*x1+y1*y1+z1*z1,1e-20f));
				subsumx+=x1*d;
				subsumy+=y1*d;
				subsumz+=z1*d;
			}
			sum_grad[threadIdx.x*3+0]=subsumx;
			sum_grad[threadIdx.x*3+1]=subsumy;
			sum_grad[threadIdx.x*3+2]=subsumz;
			for (int j=1;j<blockDim.x;j<<=1){
				__syncthreads();
				int j1=threadIdx.x;
				int j2=threadIdx.x+j;
				if ((j1&j)==0 && j2<blockDim.x){
					sum_grad[j1*3+0]+=sum_grad[j2*3+0];
					sum_grad[j1*3+1]+=sum_grad[j2*3+1];
					sum_grad[j1*3+2]+=sum_grad[j2*3+2];
				}
			}
			if (threadIdx.x==0){
				grad2[(i*m+k)*3+0]=sum_grad[0];
				grad2[(i*m+k)*3+1]=sum_grad[1];
				grad2[(i*m+k)*3+2]=sum_grad[2];
			}
			__syncthreads();
		}
	}
}
__global__ void matchcostgrad1(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,const float * __restrict__ match,float * __restrict__ grad1){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int l=threadIdx.x;l<n;l+=blockDim.x){
			float x1=xyz1[i*n*3+l*3+0];
			float y1=xyz1[i*n*3+l*3+1];
			float z1=xyz1[i*n*3+l*3+2];
			float dx=0,dy=0,dz=0;
			for (int k=0;k<m;k++){
				float x2=xyz2[i*m*3+k*3+0];
				float y2=xyz2[i*m*3+k*3+1];
				float z2=xyz2[i*m*3+k*3+2];
				float d=match[i*n*m+k*n+l]*rsqrtf(fmaxf((x1-x2)*(x1-x2)+(y1-y2)*(y1-y2)+(z1-z2)*(z1-z2),1e-20f));
				dx+=(x1-x2)*d;
				dy+=(y1-y2)*d;
				dz+=(z1-z2)*d;
			}
			grad1[i*n*3+l*3+0]=dx;
			grad1[i*n*3+l*3+1]=dy;
			grad1[i*n*3+l*3+2]=dz;
		}
	}
}
void matchcostgradLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,const float * match,float * grad1,float * grad2){
	matchcostgrad1<<<32,512>>>(b,n,m,xyz1,xyz2,match,grad1);
	matchcostgrad2<<<dim3(32,32),256>>>(b,n,m,xyz1,xyz2,match,grad2);
}

